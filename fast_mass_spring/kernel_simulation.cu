#include "hip/hip_runtime.h"


#include <cuda_device_runtime_api.h>

#include <stdio.h>

__global__ void kernel_simulation(void) {

}

//Runs the simulation need to fix the <<< issue
extern "C" void
execute_simulation() {
	printf("Running on GPU");

	kernel_simulation<<<1,1>>>();

	hipDeviceSynchronize();
}

